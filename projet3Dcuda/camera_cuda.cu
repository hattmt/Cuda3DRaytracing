#include "hip/hip_runtime.h"

#include "cuda_test.h"
#include "vec3.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include "cameras.h"

__device__ double max_min(double a, double b)
{
	if (a >= b)
		return a;
	else
		return b;
}

__device__ double abs_val(double a)
{
	if (a >= 0)
		return a;
	else
		return -a;

}



__device__  test spheretest(vec3 position, vec3 origine, vec3 direction, double r) {
	double b, a, c, delta, x1, x2, X0, Y0, Z0, t;
	test tmp;

	X0 = (origine.x - position.x);
	Y0 = (origine.y - position.y);
	Z0 = (origine.z - position.z);

	a = direction.x * direction.x + direction.y * direction.y + direction.z * direction.z;
	b = 2 * (direction.x * X0 + direction.y * Y0 + direction.z * Z0);
	c = (X0 * X0 + Y0 * Y0 + Z0 * Z0) - (r) * (r);

	delta = b * b - 4 * a * c;

	if (delta < 0)
	{
		tmp.verified = false;
		return tmp;
	}

	x1 = (-b + sqrt(delta)) / (2.0*a);
	x2 = (-b - sqrt(delta)) / (2.0*a);

	
	

		if (x2 >= 0 && x1 >= 0)
		{
			if (x2 > x1)
				t = x1;
			else
				t = x2;

			tmp.verified = true;
			tmp.value = t;
			return tmp;
		}
		


			tmp.verified = false;
			return tmp;
		

}





__device__  test triangle_MOLLER_TRUMBORE(vec3 origine, vec3 direction, vec3 A, vec3 B, vec3 C) {

	vec3 AB = sub3(B, A);//utilisation de l aire du parallelogramme ||u^v|| et des barycentres
	vec3 AC = sub3(C, A);

	float u, v, t;

	test result;

	vec3 pvec = cross3(direction, AC);
	float  det = dot3(AB, pvec);

	if (det < 0.01)
	{
		result.verified = false;
		return result;
	}

	if (abs_val(det) < 0.01)
	{
		result.verified = false;
		return result;
	}

	float invDet = 1 / det;

	vec3 tvec = sub3(origine, A);
	u = dot3(tvec, pvec) * invDet;
	if (u < 0 || u>1)
	{
		result.verified = false;
		return result;
	}

	vec3 qvec = cross3(tvec, AB);
	v = dot3(direction, qvec) * invDet;
	if (v < 0 || u + v>1)
	{
		result.verified = false;
		return result;
	}

	t = dot3(AC, qvec) * invDet;

	result.value = t;
	result.verified = true;
	return result;

}




__device__  test plan_test(vec3 position, vec3 normale, vec3 origine, vec3 direction) {

	test tmp;

	float denom = dot3(normale, direction);

	if (denom > 0.001) {

		tmp.value = dot3(sub3(position, origine), normale) / denom;
		if (tmp.value >= 0)
			tmp.verified = true;
		else
			tmp.verified = false;
	}
	else
		tmp.verified = false;

	tmp.normal = normale;

	return tmp;
}


__device__ test  test_obj_vertex(vec3 origine, vec3 direction, obj objet) {

	test result;
	vec3 A, B, C;
	double min = 1000;
	int obj = -1;
	
	for (int i = 0; i < objet.f; i++)
	{
		result = triangle_MOLLER_TRUMBORE(origine, direction, objet.vertices[i].B, objet.vertices[i].A, objet.vertices[i].C);
		
		
		
		if (result.verified)
		{
			if (result.value < min)
			{
				min = result.value;
				obj = i;
			}		
		}


	}
	if (obj != -1)
	{
		result.value = min;
		result.normal = objet.vertices[obj + objet.f].A;
		result.verified = true;
		return result;

	}
	else {
	result.verified = false;
	return result;
}

}




__device__ vec3 lancer_rayon_cuda(vec3 direction, vec3 origine, obj obj_file,scene_object* objets,int nb_objts,textur sol_bmp,int width,int height)
{
	vec3 rgb = vec3create(0, 0, 0);
	vec3 rgb_sphere = vec3create(255, 0, 0);
	

	vec3 rgb2 = vec3create(0, 0, 0);
	vec3 position_impact, surf_norml, Hv_med;
	double atten,n;
	double spec, diffus;
	float tmp, tmp1;
	vec3 L;
	char lum = 2;
	vec3 Lumiere[2] = { objets[0].position,vec3create(11.46,7.77,-9.7) };


		int obj=-1;
		double min=1000;
		test test_result;


		
	


		for (int i = 0; i < 2; i++)
		{
			for (int j = 1; j < nb_objts; j++)
			{
				if(sol_bmp.height1 == -1)
					test_result = test_obj_vertex(origine, direction, obj_file);
				else {
					if (objets[j].type == 0)
						test_result = spheretest(objets[j].position, origine, direction, objets[j].r);


					if (objets[j].type == 1)
						test_result = plan_test(objets[j].position, objets[j].normal, origine, direction); 
				}

				if (test_result.verified)
				{

					if (test_result.value < min)
					{
						min = test_result.value;
						obj = j;
					}
				}
			}

			if(obj!=-1){

					position_impact = add3(mul3(direction, min), origine);

					if (sol_bmp.height1 == -1) {
						surf_norml = test_result.normal;
					}
					else {
						if(objets[obj].type==0)
							surf_norml = normalize3(sub3(position_impact, objets[obj].position));

						if (objets[obj].type == 1)
							surf_norml = objets[obj].normal;

					}

					for (int j = 0; j < lum; j++) {
						L = sub3(Lumiere[j], position_impact);
						atten = norm3(L);
						Hv_med = normalize3(sub3(L, surf_norml));
						n = 10;
						spec = pow(dot3(surf_norml, Hv_med), n);

						diffus = dot3(surf_norml, L);

						int pixl;

						if (objets[obj].type == 1 && obj==5) {
									pixl = (position_impact.x * (position_impact.z+20)) + (position_impact.y * (position_impact.z - 20))* sol_bmp.width1/80.0;
									
									if (pixl*3 < sol_bmp.height1 * sol_bmp.width1 && pixl >= 0)
									{
										rgb = add3(mul3(vec3create(sol_bmp.pixls1[pixl * 3], sol_bmp.pixls1[pixl * 3 + 1], sol_bmp.pixls1[pixl * 3 + 2]), max_min((objets[obj].material.x + diffus * objets[obj].material.y + objets[obj].material.z * spec) * (1 / atten * 2), 0)), rgb);
									}
						}
						else
							if (objets[obj].type == 1 && obj == 6)
							{	
								tmp = (int)(position_impact.x*135) % ((int)(sol_bmp.width2/3.0));
								tmp1= (int)(position_impact.y*135 ) % ((int)(sol_bmp.height2/3.0));
								
								pixl = (tmp*3 + tmp1 * sol_bmp.width2*3);

								if (pixl  < sol_bmp.height2 * sol_bmp.width2 && pixl >= 0)
								{
									rgb = add3(mul3(vec3create(sol_bmp.pixls2[pixl ], sol_bmp.pixls2[pixl + 1], sol_bmp.pixls2[pixl  + 2]), max_min((objets[obj].material.x + diffus * objets[obj].material.y + objets[obj].material.z * spec) * (1 / atten * 2), 0)), rgb);
								}
									
							}
						else
						rgb = add3(mul3(objets[obj].rgb, max_min((objets[obj].material.x + diffus * objets[obj].material.y + objets[obj].material.z * spec) * (1 / atten), 0)), rgb);
					}

			}
			origine = position_impact;
			direction = normalize3(reflect3(position_impact, surf_norml));
		}
		

		return div3(rgb,4);

}


__device__ void pixl_cuda(UINT32* data, UINT32 width, UINT32 height, UINT32 x, UINT32 y, UINT32* rgb, UINT32 a)//AFFICHER LES PIXELS AVEC WIN32
{

	if (rgb[0] > 255)
		rgb[0] = 255;

	if (rgb[1] > 255)
		rgb[1] = 255;

	if (rgb[2] > 255)
		rgb[2] = 255;

	if (rgb[0] < 0)
		rgb[0] = 0;

	if (rgb[1] < 0)
		rgb[1] = 0;

	if (rgb[2] < 0)
		rgb[2] = 0;

	UINT32 pixl = x + y * width;

	if (height * width > pixl) {
		UINT32 color = (((UINT32)rgb[0]) << 16) + (((UINT32)rgb[1]) << 8) + ((UINT32)rgb[2]) + ((UINT32)a << 24);
		data[pixl] = color;
	}
}



__global__ void Kernel(UINT32* Pd, int width,int height,float ratioH,float ratioW,float ratioecr,vec3 position,vec3 direction,obj fichiers_obj,scene_object* objets,int nbobj,textur sol_bmp) {

	// Calculate the column index of the Pd element, denote by x
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	// Calculate the row index of the Pd element, denote by y
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	vec3 RGB,rayon;
	
	if (x < width && y < height) {
	

		double Rayon_x, Rayon_y;//coordonn�es du Rayon en fonction des coordonnees du pixels de l ecran (i,j)

		Rayon_x = (2 * ((x + 0.5) / (float)width) - 1) * ratioW;//on ramene l ecran taille (-1 ,1)(-1,1) et on recupere les vecteurs par rapport a l angle de vision en degr�s
		Rayon_y = (1 - 2 * ((y + 0.5) / (float)height) * ratioH);

		rayon = normalize3(add3(vec3create(Rayon_x, Rayon_y, -1),direction));
	

		UINT32 rgb[3] = { 0 };

		RGB= lancer_rayon_cuda( rayon, position, fichiers_obj,objets,nbobj,sol_bmp,width,height);
		
		for (int i = 0; i < 3; i++)
			rgb[i] = RGB.data[i];

		pixl_cuda(Pd, width, height, x, y, rgb, 0);
	}

}







void Camera_cuda(cameras camera,UINT32* Pd) {

	int size = camera.width * camera.height * sizeof(UINT32);
	

	float angle = 90;

	double height = camera.height;
	double width = camera.width;


	Vector3D rgb;

	float ratioecran = camera.width / (float)camera.height;// ratio taille d ecran
	float ratioW = tan(angle / (float)2 * PI / (float)180) * ratioecran;//ratio largeur
	float ratioH = tan(angle / (float)2 * PI / (float)180);//ratio hauteur


	dim3 dimBlock(16, 16);

	dim3 dimGrid((camera.width + dimBlock.x - 1) / dimBlock.x,
			(camera.height + dimBlock.y - 1) / dimBlock.y);


	vec3 position=camera.position.to_vec3cuda();
	vec3 direction = camera.direction.to_vec3cuda();
	obj fichier_obj = *camera.fichiers_obj;
	scene_object *objets_scene = camera.item;
	scene_object* obj_cuda = NULL;
	textur sol_texture = camera.bmp_sol;

	if (camera.gravity) {

		for (int i = 0; i < camera.items; i++) {
			if (objets_scene[i].type == 0) {
				objets_scene[i].speed = objets_scene[i].speed_next;
				objets_scene[i].position = objets_scene[i].position_next;
			}
		}
	}

	hipMalloc((void**)&obj_cuda, sizeof(scene_object) * camera.items);
	hipMemcpy(obj_cuda, objets_scene, sizeof(scene_object) * camera.items, hipMemcpyHostToDevice);


	Kernel << <dimGrid, dimBlock >> > ( Pd, width,  height, ratioH, ratioW, ratioecran,position,direction, fichier_obj, obj_cuda, camera.items,sol_texture);

	// transfer P from device     
	hipMemcpy(camera.data, Pd, size, hipMemcpyDeviceToHost);
	hipFree(obj_cuda);

	

	// destroy events to free memory
	/*hipEventDestroy(start);
	hipEventDestroy(stop);*/
}