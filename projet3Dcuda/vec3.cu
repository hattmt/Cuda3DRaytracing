#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <math.h>
#include "vec3.h"

__device__ vec3 vec3create(double x, double y, double z)
{
	vec3 v_tmp;
	v_tmp.x = x;
	v_tmp.y = y;
	v_tmp.z = z;
	return v_tmp;
}

 vec3 vec3create_(double x, double y, double z)
{
	vec3 v_tmp;
	v_tmp.x = x;
	v_tmp.y = y;
	v_tmp.z = z;
	return v_tmp;
}

__device__ vec3 add3(vec3 v1, vec3 v2)
{
	vec3 v_tmp;
	v_tmp.x = v1.x + v2.x;
	v_tmp.y = v1.y + v2.y;
	v_tmp.z = v1.z + v2.z;
	return v_tmp;
}

 vec3 add3_(vec3 v1, vec3 v2)
{
	vec3 v_tmp;
	v_tmp.x = v1.x + v2.x;
	v_tmp.y = v1.y + v2.y;
	v_tmp.z = v1.z + v2.z;
	return v_tmp;
}

__device__ vec3 sub3(vec3 v1, vec3 v2)
{
	vec3 v_tmp2;
	v_tmp2.x = v1.x - v2.x;
	v_tmp2.y = v1.y - v2.y;
	v_tmp2.z = v1.z - v2.z;
	return v_tmp2;
}

vec3 sub3_(vec3 v1, vec3 v2)
{
	vec3 v_tmp2;
	v_tmp2.x = v1.x - v2.x;
	v_tmp2.y = v1.y - v2.y;
	v_tmp2.z = v1.z - v2.z;
	return v_tmp2;
}

__device__ vec3 mul3(vec3 v1, double nmbr)
{
	vec3 v_tmp;

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] * nmbr;

	return v_tmp;
}

vec3 mul3_(vec3 v1, double nmbr)
{
	vec3 v_tmp;

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] * nmbr;

	return v_tmp;
}

__device__ vec3 mul3_vec(vec3 v1, vec3 v2)
{
	vec3 v_tmp;

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] * v2.data[i];

	return v_tmp;
}

__device__ vec3 div3(vec3 v1, double nmbr)
{
	vec3 v_tmp;

	if (!nmbr)
		return v_tmp;

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] / nmbr;

	return v_tmp;
}
 vec3 div3_(vec3 v1, double nmbr)
{
	vec3 v_tmp;

	if (!nmbr)
		return v_tmp;

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] / nmbr;

	return v_tmp;
}

__device__ double norm3(vec3 v1)
{
	return sqrt(v1.x * v1.x + v1.y * v1.y + v1.z * v1.z);
}

 double norm3_(vec3 v1)
{
	return sqrt(v1.x * v1.x + v1.y * v1.y + v1.z * v1.z);
}


__device__ vec3 normalize3(vec3 v1)
{
	vec3 v_tmp;
	double normv1 = norm3(v1);

	if (!normv1)
		return vec3create(0,0,0);

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] / normv1;

	return v_tmp;
}

vec3 normalize3_(vec3 v1)
{
	vec3 v_tmp;
	double normv1 = norm3_(v1);

	if (!normv1)
		return vec3create_(0, 0, 0);

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] / normv1;

	return v_tmp;
}

__device__ double distance3(vec3 v1, vec3 v2)
{
	vec3 v_tmp = sub3(v2, v1);
	return norm3(v_tmp);
}

 double distance3_(vec3 v1, vec3 v2)
{
	vec3 v_tmp = sub3_(v2, v1);
	return norm3_(v_tmp);
}

__device__ double dot3(vec3 v1, vec3 v2)
{
	vec3 v_tmp;

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] * v2.data[i];

	return v_tmp.x + v_tmp.y + v_tmp.z;
}

double dot3_(vec3 v1, vec3 v2)
{
	vec3 v_tmp;

	for (int i = 0; i < 3; i++)
		v_tmp.data[i] = v1.data[i] * v2.data[i];

	return v_tmp.x + v_tmp.y + v_tmp.z;
}

__device__ vec3 cross3(vec3 v1, vec3 v2)
{
	vec3 v_tmp;
	v_tmp.x = v1.y * v2.z - v1.z * v2.y;
	v_tmp.y = v1.z * v2.x - v1.x * v2.z;
	v_tmp.z = v1.x * v2.y - v1.y * v2.x;
	return v_tmp;
}

__device__ vec3 reflect3(vec3 v, vec3 vnormal)
{
	return sub3(v, mul3(vnormal, dot3(v, vnormal) * 2));
}

vec3 reflect3_(vec3 v, vec3 vnormal)
{
	return sub3_(v, mul3_(vnormal, dot3_(v, vnormal) * 2));
}

__device__ vec3 set3(double x, double y, double z)
{
	vec3 v;
	v.x = x;
	v.y = y;
	v.z = z;
	return v;
}

__device__ vec3 copy3(vec3 v)
{
	vec3 copy;
	copy.x = v.x;
	copy.y = v.y;
	copy.z = v.z;
	return copy;
}


